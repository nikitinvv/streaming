#include "hip/hip_runtime.h"
#include "radonortho.cuh"
#include "kernels.cuh"
#include  <stdio.h>
radonortho::radonortho(size_t ntheta, size_t n, size_t nz)
 : ntheta(ntheta), n(n), nz(nz) 
{
	// arrays allocation on GPU
	hipMalloc((void **)&fx, n * nz * sizeof(float));
	hipMalloc((void **)&fy, n * nz * sizeof(float));
	hipMalloc((void **)&fz, n * n * sizeof(float));
	hipMalloc((void **)&g, n * ntheta * nz * sizeof(float));
	hipMalloc((void **)&gs, n * ntheta * nz * sizeof(unsigned char));	
	hipMalloc((void **)&flat, n * nz * sizeof(unsigned char));
	
	hipMalloc((void **)&fg, (n / 2 + 1) * ntheta * nz * sizeof(float2));
	hipMalloc((void **)&filter, (n / 2 + 1) * sizeof(float));	
	hipMalloc((void **)&theta, ntheta * sizeof(float));

	hipMemset(fx,0,n*nz*sizeof(float));
	hipMemset(fy,0,n*nz*sizeof(float));
	hipMemset(fz,0,n*n*sizeof(float));
	
	//fft plans for filtering
	int ffts[] = {n};
	int idist = n;
	int odist = n / 2 + 1;
	int inembed[] = {n};
	int onembed[] = {n / 2 + 1};
	hipfftPlanMany(&plan_forward, 1, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, ntheta * nz);
	hipfftPlanMany(&plan_inverse, 1, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, ntheta * nz);

	//init thread blocks and block grids
	BS3d.x = 32;
	BS3d.y = 32;
	BS3d.z = 1;

	GS3d1.x = ceil(n / (float)BS3d.x);
	GS3d1.y = ceil(ntheta / (float)BS3d.y);
	GS3d1.z = ceil(nz / (float)BS3d.z);

	GS3d2.x = ceil(n / (float)BS3d.x);
	GS3d2.y = ceil(n / (float)BS3d.y);
	
	GS3d3.x = ceil(n / (float)BS3d.x);
	GS3d3.y = ceil(nz / (float)BS3d.y);

	is_free = false;	
}


// destructor, memory deallocation
radonortho::~radonortho() { free(); }


void radonortho::free()
{
	if (!is_free) 
	{
		hipFree(g);
		hipFree(gs);		
		hipFree(fg);
		hipFree(fx);
		hipFree(fy);
		hipFree(fz);
		hipFree(filter);

		hipFree(theta);
		hipfftDestroy(plan_forward);
		hipfftDestroy(plan_inverse);
		is_free = true;   
	}
	
}


void radonortho::rec(size_t fx_,size_t fy_,size_t fz_, size_t g_, size_t theta_, float center, int ix, int iy, int iz)
{
	// copy data and angles to GPU
	hipMemcpy(gs, (unsigned char *)g_, n * ntheta * nz * sizeof(unsigned char), hipMemcpyDefault);	
	hipMemcpy(theta, (float *)theta_, ntheta * sizeof(float), hipMemcpyDefault);
	
	//hipMemcpy(g, (float *)g_, n * ntheta * nz * sizeof(float), hipMemcpyDefault);	
	
	// convert short to float
	correction<<<GS3d1, BS3d>>>(g, gs, flat, n, ntheta, nz);	
	

	// fft for filtering in the frequency domain
	hipfftExecR2C(plan_forward, (hipfftReal *)g, (hipfftComplex *)fg);
	// parzen filtering
	applyfilter<<<GS3d1, BS3d>>>(fg, filter, n, ntheta, nz);
	// fft back
	hipfftExecC2R(plan_inverse, (hipfftComplex *)fg, (hipfftReal *)g);
	//hipMemcpy((float *)g_, g, n * ntheta * nz * sizeof(float), hipMemcpyDefault);	
	
	// reconstruct slices via summation over lines	
	ortho_kerx<<<GS3d3, BS3d>>>(fx, g, theta, center, ix, n, ntheta, nz);
	ortho_kery<<<GS3d3, BS3d>>>(fy, g, theta, center, iy, n, ntheta, nz);	
	ortho_kerz<<<GS3d2, BS3d>>>(fz, g, theta, center, iz, n, ntheta, nz);
	
	//copy result to cpu
	hipMemcpy((float *)fx_, fx, n * nz * sizeof(float), hipMemcpyDefault);
	hipMemcpy((float *)fy_, fy, n * nz * sizeof(float), hipMemcpyDefault);
	hipMemcpy((float *)fz_, fz, n * n * sizeof(float), hipMemcpyDefault);
}

void radonortho::set_filter(size_t filter_)
{
	hipMemcpy(filter, (float*) filter_, (n/2+1)*sizeof(float),hipMemcpyDefault);
}

void radonortho::set_flat(size_t flat_)
{
	hipMemcpy(flat, (unsigned char*) flat_, n*nz*sizeof(unsigned char),hipMemcpyDefault);
	
}